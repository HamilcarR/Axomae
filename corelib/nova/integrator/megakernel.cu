#include "hip/hip_runtime.h"
#include "DrawEngine.h"
#include "GPUIntegrator.cuh"
#include "Integrator.h"
#include "engine/nova_exception.h"
#include "internal/common/math/math_texturing.h"
#include "internal/debug/Logger.h"
#include "internal/device/gpgpu/cuda/CudaDevice.h"
#include "internal/device/gpgpu/device_transfer_interface.h"
#include "internal/device/gpgpu/device_utils.h"
#include "internal/device/gpgpu/kernel_launch_interface.h"
#include "manager/NovaResourceManager.h"

namespace resrc = device::gpgpu;

/* Serves only as a baseline for performance to compare against */
namespace nova {
  namespace gpu {
    AX_KERNEL void test_func(float *ptr, hipTextureObject_t host_texture, unsigned width, unsigned height, int i_width, int i_height) {
      unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
      unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
      unsigned int offset = (y * width + x) * 4;

      if (offset < width * height * 4) {
        float u = math::texture::pixelToUv(x, width);
        float v = math::texture::pixelToUv(y, height);

        float4 sample{};
        sample = tex2D<float4>(host_texture, u, v);
        ptr[offset] = sample.x;
        ptr[offset + 1] = sample.y;
        ptr[offset + 2] = sample.z;
        ptr[offset + 3] = 1.f;
      }
    }
  }  // namespace gpu

  static void setup_descriptors(resrc::texture_descriptor &tex_desc, resrc::resource_descriptor &resrc_desc) {
    resrc::channel_format &ch_format = tex_desc.channel_descriptor;
    ch_format.bits_size_x = 32;
    ch_format.bits_size_y = 32;
    ch_format.bits_size_z = 32;
    ch_format.bits_size_a = 32;
    ch_format.format_type = resrc::FLOAT;

    tex_desc.filter_mode = resrc::FILTER_LINEAR;
    tex_desc.read_mode = resrc::READ_ELEMENT_TYPE;
    tex_desc.address_mode[0] = tex_desc.address_mode[1] = resrc::ADDRESS_BORDER;
    tex_desc.normalized_coords = true;
    resrc_desc.resource_buffer_descriptors.res.array.array = nullptr;
    resrc_desc.type = resrc::RESOURCE_ARRAY;
  }

  void launch_gpu_kernel(HdrBufferStruct *buffers,
                         unsigned screen_width,
                         unsigned screen_height,
                         NovaRenderEngineInterface *engine_interface,
                         nova::nova_eng_internals &nova_internals) {

    if (ax_cuda::utils::cuda_info_device().empty()) {
      LOGS("No suitable gpu detected.");
      nova_internals.exception_manager->addError(nova::exception::GENERAL_GPU_ERROR);
      return;
    }

    const NovaResourceManager *resource_manager = nova_internals.resource_manager;
    const texturing::TextureRawData image_texture = resource_manager->getEnvmapData();
    std::size_t screen_size = screen_width * screen_height * buffers->channels * sizeof(float);

    resrc::texture_descriptor tex_desc{};
    resrc::resource_descriptor res_desc{};
    setup_descriptors(tex_desc, res_desc);
    resrc::GPU_texture texture_resrc = resrc::create_texture(
        (const void *)image_texture.raw_data, image_texture.width, image_texture.height, tex_desc, res_desc);
    resrc::GPU_query_result draw_buffer = resrc::allocate_buffer(screen_size);
    AXCUDA_ERROR_CHECK(draw_buffer.error_status);

    kernel_argpack_t argpack;
    argpack.num_blocks = {screen_width / 32, screen_height, 1};
    argpack.block_size = {32, 1, 1};
    exec_kernel(argpack,
                gpu::test_func,
                (float *)draw_buffer.device_ptr,
                std::any_cast<hipTextureObject_t>(texture_resrc.texture_object),
                screen_width,
                screen_height,
                image_texture.width,
                image_texture.height);
    AXCUDA_ERROR_CHECK(resrc::copy_buffer(draw_buffer.device_ptr, buffers->partial_buffer, screen_size, 1).error_status);
    AXCUDA_ERROR_CHECK(resrc::deallocate_buffer(draw_buffer.device_ptr).error_status);
    resrc::destroy_texture(texture_resrc);
  }
}  // namespace nova
