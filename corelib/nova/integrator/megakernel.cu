#include "hip/hip_runtime.h"
#include "DrawEngine.h"
#include "GPUIntegrator.cuh"
#include "Integrator.h"
#include "engine/nova_exception.h"
#include "internal/debug/Logger.h"
#include "internal/debug/PerformanceLogger.h"
#include "internal/device/gpgpu/cuda/CudaDevice.h"
#include "internal/device/gpgpu/device_resource_interface.h"
#include "internal/device/gpgpu/device_utils.h"
#include "internal/device/gpgpu/kernel_launch_interface.h"
#include "manager/NovaResourceManager.h"
#include "math/math_texturing.h"

/* Serves only as a baseline for performance to compare against */
namespace nova {
  namespace gpu {
    AX_KERNEL void test_func(float *ptr, hipTextureObject_t host_texture, unsigned width, unsigned height, int i_width, int i_height) {
      unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
      unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
      unsigned int offset = (y * width + x) * 4;

      if (offset < width * height * 4) {
        float u = math::texture::pixelToUv(x, width);
        float v = math::texture::pixelToUv(y, height);

        float4 sample{};
        sample = tex2D<float4>(host_texture, u, v);
        ptr[offset] = sample.x;
        ptr[offset + 1] = sample.y;
        ptr[offset + 2] = sample.z;
        ptr[offset + 3] = 1.f;
      }
    }
  }  // namespace gpu
  void launch_gpu_kernel(HdrBufferStruct *buffers,
                         unsigned screen_width,
                         unsigned screen_height,
                         NovaRenderEngineInterface *engine_interface,
                         nova::nova_eng_internals &nova_internals) {
    if (ax_cuda::utils::cuda_info_device().empty()) {
      LOGS("No suitable gpu detected.");
      nova_internals.exception_manager->addError(nova::exception::GENERAL_GPU_ERROR);
      return;
    }
    namespace resrc = device::gpgpu::resource;
    const NovaResourceManager *resource_manager = nova_internals.resource_manager;
    const texturing::TextureRawData image_texture = resource_manager->getEnvmapData();
    std::size_t screen_size = screen_width * screen_height * buffers->channels * sizeof(float);
    resrc::texture_channel_descriptor desc{};
    desc.bits_size_x = 32;
    desc.bits_size_y = 32;
    desc.bits_size_z = 32;
    desc.bits_size_a = 32;
    desc.texture_type = resrc::FLOAT;
    resrc::GPU_texture texture_resrc = resrc::create_texture((const void *)image_texture.raw_data, image_texture.width, image_texture.height, desc);
    resrc::GPU_resource draw_buffer = resrc::allocate_buffer(screen_size);
    AXCUDA_ERROR_CHECK(draw_buffer.error_status);

    kernel_argpack_t argpack;
    argpack.num_blocks = {screen_width / 32, screen_height, 1};
    argpack.block_size = {32, 1, 1};
    exec_kernel(argpack,
                gpu::test_func,
                (float *)draw_buffer.device_ptr,
                std::any_cast<hipTextureObject_t>(texture_resrc.texture_object),
                screen_width,
                screen_height,
                image_texture.width,
                image_texture.height);
    AXCUDA_ERROR_CHECK(resrc::copy_buffer(draw_buffer.device_ptr, buffers->partial_buffer, screen_size, 1).error_status);
    AXCUDA_ERROR_CHECK(resrc::deallocate_buffer(draw_buffer.device_ptr).error_status);
    resrc::destroy_texture(texture_resrc);
  }
}  // namespace nova
