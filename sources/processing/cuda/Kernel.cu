#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include <cmath>
#include <sstream>

// Old code , tb refactored

namespace axomae {

  AX_DEVICE_ONLY const static bool isbigEndian = SDL_BIG_ENDIAN == SDL_BYTEORDER;
  AX_DEVICE_ONLY static uint32_t max_int_rgb = 0;
  AX_DEVICE_ONLY static uint32_t min_int_rgb = UINT32_MAX;

  struct Triplet {
    int x;
    int y;
    int z;
  };

  struct DEVICE_RGB {
   public:
    uint8_t r;
    uint8_t g;
    uint8_t b;
    uint8_t a;

    AX_DEVICE_ONLY void operator=(DEVICE_RGB rgb) {
      this->r = rgb.r;
      this->g = rgb.g;
      this->b = rgb.b;
      this->a = rgb.a;
    }
    std::string string() { return std::to_string(r) + "   " + std::to_string(g) + "  " + std::to_string(b) + "\n"; }
    AX_DEVICE_ONLY DEVICE_RGB operator+(DEVICE_RGB rgb) { return {uint8_t(r + rgb.r), uint8_t(g + rgb.g), uint8_t(b + rgb.b), uint8_t(a + rgb.a)}; }
    template<typename T>
    AX_DEVICE_ONLY DEVICE_RGB operator+(T rgb) {
      return {r + rgb, g + rgb, b + rgb, a + rgb};
    }
    AX_DEVICE_ONLY DEVICE_RGB operator*(DEVICE_RGB rgb) { return {uint8_t(r * rgb.r), uint8_t(g * rgb.g), uint8_t(b * rgb.b), uint8_t(a * rgb.a)}; }
    template<typename T>
    AX_DEVICE_ONLY DEVICE_RGB operator*(T value) {
      return {uint8_t(r * value), uint8_t(g * value), uint8_t(b * value), uint8_t(a * value)};
    }
    AX_DEVICE_ONLY DEVICE_RGB normalize_rgb(DEVICE_RGB max, DEVICE_RGB min) {
      uint8_t n_red = normalize(max.r, min.r, r);
      uint8_t n_green = normalize(max.g, min.g, g);
      uint8_t n_blue = normalize(max.b, min.b, b);
      return {n_red, n_green, n_blue, 0};
    }
    /*compute the magnitude between to rgb values*/
    AX_DEVICE_ONLY DEVICE_RGB magnitude_rgb(DEVICE_RGB horizontal, DEVICE_RGB vertical) {
      DEVICE_RGB rgb;
      rgb.r = (uint8_t)magnitude(vertical.r, horizontal.r);
      rgb.g = (uint8_t)magnitude(vertical.g, horizontal.g);
      rgb.b = (uint8_t)magnitude(vertical.b, horizontal.b);
      rgb.a = (uint8_t)magnitude(vertical.a, horizontal.a);
      return rgb;
    }
    AX_DEVICE_ONLY void print() { printf("%i %i %i\n", r, g, b); }
  };

  class SDLSurfParam {
   public:
    unsigned int width;
    unsigned int height;
    int bpp;
    int pitch;
    void *data;
    SDLSurfParam(SDL_Surface *im) {
      width = im->w;
      height = im->h;
      bpp = im->format->BytesPerPixel;
      pitch = im->pitch;
      data = im->pixels;
    }
    SDLSurfParam() {}
    size_t getByteSize() { return height * pitch; }
  };
  template<typename T>
  struct custom_convolution_kernel {
    T *array;
    uint8_t size_w;
    uint8_t size_h;
  };

  /*device*/
  /*********************************************************************************************************************************************/
  AX_DEVICE_CALLABLE uint32_t rgb_to_int(DEVICE_RGB val) {
    uint32_t value = (isbigEndian) ? val.a | (val.b << 8) | (val.g << 16) | (val.r << 24) : val.r | (val.g << 8) | (val.b << 16) | (val.a << 24);
    return value;
  }
  AX_DEVICE_ONLY void initialize_2D_array(uint32_t *array, int size_w, int size_h) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    array[i * size_w + j] = 0;
  }
  AX_DEVICE_ONLY DEVICE_RGB compute_greyscale(DEVICE_RGB rgb, const bool luminance) {
    DEVICE_RGB ret;
    if (luminance) {
      ret.r = rgb.r * 0.3 + rgb.g * 0.59 + rgb.b * 0.11;
      ret.g = ret.r;
      ret.b = ret.r;
    } else {
      ret.r = (int)((rgb.r + rgb.b + rgb.g) / 3);
      ret.g = ret.r;
      ret.b = ret.r;
    }
    return ret;
  }
  AX_DEVICE_ONLY DEVICE_RGB int_to_rgb(uint8_t *pixel_value, const int bpp) {
    DEVICE_RGB rgb = {0, 0, 0, 0};
    if (bpp == 4) {
      if (isbigEndian) {
        rgb.r = *pixel_value >> 24 & 0xFF;
        rgb.g = *pixel_value >> 16 & 0xFF;
        rgb.b = *pixel_value >> 8 & 0xFF;
        rgb.a = *pixel_value & 0xFF;
      } else {
        rgb.a = *pixel_value >> 24 & 0xFF;
        rgb.b = *pixel_value >> 16 & 0xFF;
        rgb.g = *pixel_value >> 8 & 0xFF;
        rgb.r = *pixel_value & 0xFF;
      }

    } else if (bpp == 3) {
      if (isbigEndian) {
        rgb.r = pixel_value[0];
        rgb.g = pixel_value[1];
        rgb.b = pixel_value[2];
        rgb.a = 0;
      } else {
        rgb.b = pixel_value[0];
        rgb.g = pixel_value[1];
        rgb.r = pixel_value[2];
        rgb.a = 0;
      }

    } else if (bpp == 2) {
      if (isbigEndian) {
        rgb.r = *pixel_value >> 12 & 0xF;
        rgb.g = *pixel_value >> 8 & 0XF;
        rgb.b = *pixel_value >> 4 & 0XF;
        rgb.a = *pixel_value & 0XF;
      } else {
        rgb.a = *pixel_value >> 12 & 0xF;
        rgb.b = *pixel_value >> 8 & 0XF;
        rgb.g = *pixel_value >> 4 & 0XF;
        rgb.r = *pixel_value & 0XF;
      }
    } else if (bpp == 1) {
      if (isbigEndian) {
        rgb.r = *pixel_value >> 5 & 0X7;
        rgb.g = *pixel_value >> 2 & 0X7;
        rgb.b = *pixel_value & 0X3;
        rgb.a = 0;
      } else {
        rgb.b = *pixel_value >> 5 & 0X7;
        rgb.g = *pixel_value >> 2 & 0X7;
        rgb.r = *pixel_value & 0X3;
        rgb.a = 0;
      }
    }
    return rgb;
  }
  AX_DEVICE_ONLY void set_pixel_color(uint8_t *pixel_value, DEVICE_RGB rgb, const int bpp) {
    uint32_t toInt = rgb_to_int(rgb);
    if (bpp == 4)
      *(uint32_t *)(pixel_value) = toInt;
    else if (bpp == 3) {
      if (isbigEndian) {
        ((uint8_t *)pixel_value)[0] = toInt >> 16 & 0xFF;
        ((uint8_t *)pixel_value)[1] = toInt >> 8 & 0xFF;
        ((uint8_t *)pixel_value)[2] = toInt & 0xFF;
      } else {
        ((uint8_t *)pixel_value)[0] = toInt & 0xFF;
        ((uint8_t *)pixel_value)[1] = toInt >> 8 & 0xFF;
        ((uint8_t *)pixel_value)[2] = toInt >> 16 & 0xFF;
      }
    } else if (bpp == 2)
      *((uint16_t *)pixel_value) = toInt;
    else
      *pixel_value = toInt;
  }
  AX_DEVICE_ONLY DEVICE_RGB get_pixel_value_at(uint8_t *pixel, int i, int j, const int bpp, int pitch) {
    uint8_t *p = (uint8_t *)(pixel) + i * bpp + j * pitch;
    DEVICE_RGB A = int_to_rgb(p, bpp);
    return A;
  }
  struct convolution_directions {
    DEVICE_RGB vertical;
    DEVICE_RGB horizontal;
  };

  // TODO : case kernel < 0
  AX_DEVICE_ONLY convolution_directions compute_convolution(uint8_t *pixel,
                                                      const int bpp,
                                                      int pitch,
                                                      const int h_kernel[KERNEL_SIZE][KERNEL_SIZE],
                                                      const int v_kernel[KERNEL_SIZE][KERNEL_SIZE],
                                                      uint8_t border_flag) {
    DEVICE_RGB center = get_pixel_value_at(pixel, 0, 0, bpp, pitch);
    DEVICE_RGB west = get_pixel_value_at(pixel, 0, -1, bpp, pitch);  // here : if threadIdx.y = 0 bug
    DEVICE_RGB north_west = get_pixel_value_at(pixel, -1, -1, bpp, pitch);
    DEVICE_RGB north = get_pixel_value_at(pixel, -1, 0, bpp, pitch);
    DEVICE_RGB north_east = get_pixel_value_at(pixel, -1, 1, bpp, pitch);
    DEVICE_RGB east = get_pixel_value_at(pixel, 0, 1, bpp, pitch);
    DEVICE_RGB south_east = get_pixel_value_at(pixel, 1, 1, bpp, pitch);
    DEVICE_RGB south = get_pixel_value_at(pixel, 1, 0, bpp, pitch);
    DEVICE_RGB south_west = get_pixel_value_at(pixel, 1, -1, bpp, pitch);
    double verticalx = north_west.r * v_kernel[0][0] + north.r * v_kernel[0][1] + north_east.r * v_kernel[0][2] + west.r * v_kernel[1][0] +
                       center.r * v_kernel[1][1] + east.r * v_kernel[1][2] + south_west.r * v_kernel[2][0] + south.r * v_kernel[2][1] +
                       south_east.r * v_kernel[2][2];
    double verticaly = north_west.g * v_kernel[0][0] + north.g * v_kernel[0][1] + north_east.g * v_kernel[0][2] + west.g * v_kernel[1][0] +
                       center.g * v_kernel[1][1] + east.g * v_kernel[1][2] + south_west.g * v_kernel[2][0] + south.g * v_kernel[2][1] +
                       south_east.g * v_kernel[2][2];
    double verticalz = north_west.b * v_kernel[0][0] + north.b * v_kernel[0][1] + north_east.b * v_kernel[0][2] + west.b * v_kernel[1][0] +
                       center.b * v_kernel[1][1] + east.b * v_kernel[1][2] + south_west.b * v_kernel[2][0] + south.b * v_kernel[2][1] +
                       south_east.b * v_kernel[2][2];
    double horizontalx = north_west.r * h_kernel[0][0] + north.r * h_kernel[0][1] + north_east.r * h_kernel[0][2] + west.r * h_kernel[1][0] +
                         center.r * h_kernel[1][1] + east.r * h_kernel[1][2] + south_west.r * h_kernel[2][0] + south.r * h_kernel[2][1] +
                         south_east.r * h_kernel[2][2];
    double horizontaly = north_west.g * h_kernel[0][0] + north.g * h_kernel[0][1] + north_east.g * h_kernel[0][2] + west.g * h_kernel[1][0] +
                         center.g * h_kernel[1][1] + east.g * h_kernel[1][2] + south_west.g * h_kernel[2][0] + south.g * h_kernel[2][1] +
                         south_east.g * h_kernel[2][2];
    double horizontalz = north_west.b * h_kernel[0][0] + north.b * h_kernel[0][1] + north_east.b * h_kernel[0][2] + west.b * h_kernel[1][0] +
                         center.b * h_kernel[1][1] + east.b * h_kernel[1][2] + south_west.b * h_kernel[2][0] + south.b * h_kernel[2][1] +
                         south_east.b * h_kernel[2][2];
    convolution_directions dir;
    DEVICE_RGB minn = {0, 0, 0, 0};
    DEVICE_RGB maxx = {255, 255, 255, 255};
    uint8_t rh = (uint8_t)normalize(maxx.r, minn.r, horizontalx);
    uint8_t rv = (uint8_t)normalize(maxx.r, minn.r, verticalx);
    uint8_t gh = (uint8_t)normalize(maxx.r, minn.r, horizontaly);
    uint8_t gv = (uint8_t)normalize(maxx.r, minn.r, verticaly);
    uint8_t bh = (uint8_t)normalize(maxx.r, minn.r, horizontalz);
    uint8_t bv = (uint8_t)normalize(maxx.r, minn.r, verticalz);
    DEVICE_RGB vertical = {rv, gv, bv, 0};
    DEVICE_RGB horizontal = {rh, gh, bh, 0};
    dir.vertical = vertical;
    dir.horizontal = horizontal;
    return dir;
  }

  /* pos 0 = vertical convolution kernel
     pos 1 = horizontal convolution kernel */

  AX_DEVICE_ONLY DEVICE_RGB get_convolution_values(uint8_t *pixel, const int bpp, int pitch, uint8_t convolution, uint8_t border) {
    int custom_kernel = 0;
    convolution_directions convoluted;
    if (custom_kernel == 0) {
      if (convolution == AXOMAE_USE_SOBEL)
        convoluted = compute_convolution(pixel, bpp, pitch, sobel_mask_horizontal, sobel_mask_vertical, border);
      else if (convolution == AXOMAE_USE_PREWITT)
        convoluted = compute_convolution(pixel, bpp, pitch, prewitt_mask_horizontal, prewitt_mask_vertical, border);
      else
        convoluted = compute_convolution(pixel, bpp, pitch, scharr_mask_horizontal, scharr_mask_vertical, border);
      DEVICE_RGB var = convoluted.vertical.magnitude_rgb(convoluted.vertical, convoluted.horizontal);
      return var;
    } else {
      // TODO : add custom kernels processing
      return {0, 0, 0, 0};
    }
  }

  AX_DEVICE_ONLY DEVICE_RGB compute_normal(uint8_t *pixel, int bpp, int pitch, double factor) {
    DEVICE_RGB center = get_pixel_value_at(pixel, 0, 0, bpp, pitch);
    DEVICE_RGB west = get_pixel_value_at(pixel, 0, -1, bpp, pitch);
    DEVICE_RGB north_west = get_pixel_value_at(pixel, -1, -1, bpp, pitch);
    DEVICE_RGB north = get_pixel_value_at(pixel, -1, 0, bpp, pitch);
    DEVICE_RGB north_east = get_pixel_value_at(pixel, -1, 1, bpp, pitch);
    DEVICE_RGB east = get_pixel_value_at(pixel, 0, 1, bpp, pitch);
    DEVICE_RGB south_east = get_pixel_value_at(pixel, 1, 1, bpp, pitch);
    DEVICE_RGB south = get_pixel_value_at(pixel, 1, 0, bpp, pitch);
    DEVICE_RGB south_west = get_pixel_value_at(pixel, 1, -1, bpp, pitch);
    float dx = factor * (east.g - west.g) / 255;
    float dy = factor * (north.g - south.g) / 255;
    float ddx = factor * (north_east.g - south_west.g) / 255;
    float ddy = factor * (north_west.g - south_east.g) / 255;
    float Nx = normalize(-1, 1, lerp(dy, ddy, 0.5));
    float Ny = normalize(-1, 1, lerp(dx, ddx, 0.5));
    uint8_t Nz = 255;
    if (Nx >= 255)
      Nx = 255;
    else if (Nx <= 0)
      Nx = 0;
    if (Ny >= 255)
      Ny = 255;
    else if (Ny <= 0)
      Ny = 0;
    return {(uint8_t)std::floor(Nx), (uint8_t)std::floor(Ny), Nz, 0};
  }

  AX_DEVICE_ONLY static void replace_min(DEVICE_RGB rgb) {
    uint32_t *max = &max_int_rgb;
    uint8_t *pixel = (uint8_t *)max;
    DEVICE_RGB maxx = int_to_rgb(pixel, 4);
    maxx.r = maxx.r >= rgb.r ? maxx.r : rgb.r;
    maxx.g = maxx.g >= rgb.g ? maxx.g : rgb.g;
    maxx.b = maxx.b >= rgb.b ? maxx.b : rgb.b;
    *max = rgb_to_int(maxx);
  }

  AX_DEVICE_ONLY static void replace_max(DEVICE_RGB rgb) {
    uint32_t *min = &min_int_rgb;
    uint8_t *pixel = (uint8_t *)min;
    DEVICE_RGB minn = int_to_rgb(pixel, 4);
    minn.r = minn.r < rgb.r ? minn.r : rgb.r;
    minn.g = minn.g < rgb.g ? minn.g : rgb.g;
    minn.b = minn.b < rgb.b ? minn.b : rgb.b;
    *min = rgb_to_int(minn);
  }

  /* kernels */
  /*********************************************************************************************************************************************/
  __global__ void GPU_compute_greyscale(void *array, int size_w, int size_h, const int bpp, int pitch, const bool luminance) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < size_w && j < size_h) {
      DEVICE_RGB rgb = {0, 0, 0, 0};
      uint8_t *pixel_value = (uint8_t *)(array) + i * bpp + j * pitch;
      rgb = compute_greyscale(int_to_rgb(pixel_value, bpp), luminance);
      set_pixel_color(pixel_value, rgb, bpp);
    }
  }
  __global__ void GPU_compute_edges(
      void *image, void *save, unsigned int width, unsigned int height, int bpp, int pitch, uint8_t convolution, uint8_t border) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < width - 1 && j < height - 1 && i > 0 && j > 0) {
      uint8_t *pixel = (uint8_t *)(image) + i * bpp + j * pitch;
      uint8_t *p = (uint8_t *)(save) + i * bpp + j * pitch;
      DEVICE_RGB rgb = get_convolution_values(pixel, bpp, pitch, convolution, border);
      set_pixel_color(p, rgb, bpp);
    } else {  // just some weird cases , TODO later
      uint8_t *pixel = (uint8_t *)(image) + i * bpp + j * pitch;
      if (border == AXOMAE_REPEAT) {
      } else if (border == AXOMAE_CLAMP) {
      } else {
      }
    }
  }

  __global__ void GPU_compute_normals(
      void *image, void *save, unsigned int width, unsigned int height, int bpp, int pitch, double factor, uint8_t border) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    uint8_t *pixel = (uint8_t *)(image) + i * bpp + j * pitch;
    uint8_t *write = (uint8_t *)(save) + i * bpp + j * pitch;
    if (i < width - 1 && j < height - 1 && i > 1 && j > 1) {
      DEVICE_RGB rgb = compute_normal(pixel, bpp, pitch, factor);
      set_pixel_color(write, rgb, bpp);
    } else {
      if (border == AXOMAE_REPEAT) {
      } else if (border == AXOMAE_CLAMP) {
      } else {
      }
    }
  }

  /*host functions*/
  /*********************************************************************************************************************************************/

  void GPU_compute_greyscale(SDL_Surface *image, const bool luminance) {
    int width = image->w;
    int height = image->h;
    int pitch = image->pitch;
    int bpp = image->format->BytesPerPixel;
    void *D_image;
    size_t size = pitch * height;
    hipMalloc((void **)&D_image, size);
    hipMemcpy(D_image, image->pixels, size, hipMemcpyHostToDevice);
    gpu_threads D = get_optimal_thread_distribution(width, height);
    GPU_compute_greyscale<<<D.blocks, D.threads>>>(D_image, width, height, bpp, pitch, luminance);
    check_error(__FILE__, __LINE__);
    SDL_LockSurface(image);
    hipMemcpy(image->pixels, D_image, size, hipMemcpyDeviceToHost);
    SDL_UnlockSurface(image);
    hipFree(D_image);
  }

  void GPU_compute_height(SDL_Surface *greyscale, uint8_t convolution, uint8_t border) {
    SDLSurfParam param(greyscale);
    void *D_image, *R_image;
    size_t size = param.getByteSize();
    hipMalloc((void **)&D_image, size);
    hipMalloc((void **)&R_image, size);
    hipMemcpy(D_image, param.data, size, hipMemcpyHostToDevice);
    gpu_threads D = get_optimal_thread_distribution(param.width, param.height);
    D.blocks.x++;  // border management
    D.blocks.y++;  //
    GPU_compute_edges<<<D.blocks, D.threads>>>(D_image, R_image, param.width, param.height, param.bpp, param.pitch, convolution, border);
    check_error(__FILE__, __LINE__);
    SDL_LockSurface(greyscale);
    hipMemcpy(greyscale->pixels, R_image, size, hipMemcpyDeviceToHost);
    SDL_UnlockSurface(greyscale);
    hipFree(D_image);
    hipFree(R_image);
  }

  void GPU_compute_normal(SDL_Surface *height, double factor, uint8_t border) {

    SDLSurfParam param(height);
    void *D_image, *D_save;
    hipMalloc((void **)&D_image, param.getByteSize());
    hipMalloc((void **)&D_save, param.getByteSize());

    hipMemcpy(D_image, param.data, param.getByteSize(), hipMemcpyHostToDevice);
    gpu_threads blocks = get_optimal_thread_distribution(param.width, param.height);
    blocks.blocks.x++;
    blocks.blocks.y++;
    GPU_compute_normals<<<blocks.blocks, blocks.threads>>>(D_image, D_save, param.width, param.height, param.bpp, param.pitch, factor, border);
    check_error(__FILE__, __LINE__);
    SDL_LockSurface(height);
    hipMemcpy(height->pixels, D_save, param.getByteSize(), hipMemcpyDeviceToHost);
    SDL_UnlockSurface(height);

    hipFree(D_image);
    hipFree(D_save);
  }

};  // namespace axomae
